#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <chrono>
#include <hip/hip_runtime.h>
#include "utils.hpp"  // 确保路径正确

#define FILTER_RADIUS 2
#define SIGMA_S 10.0f
#define SIGMA_R 50.0f

__device__ float gaussian(float x, float sigma) {
    return expf(-0.5f * x * x / (sigma * sigma));
}

__device__ float d_bilateral_filter(unsigned char* image_buffer,
                                    int x, int y, int width, int height, int channels,
                                    float sigma_s, float sigma_r, int pixel_id) {
    float iFiltered = 0.0f;
    float wP = 0.0f;
    
    for (int dy = -FILTER_RADIUS; dy <= FILTER_RADIUS; dy++) {
        for (int dx = -FILTER_RADIUS; dx <= FILTER_RADIUS; dx++) {
            int nx = x + dx;
            int ny = y + dy;

            if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
                int neighbor_id = (ny * width + nx) * channels + pixel_id % channels;
                float dist = sqrtf(dx * dx + dy * dy);
                float colorDistance = fabs(float(image_buffer[neighbor_id]) - float(image_buffer[pixel_id]));

                float gs = gaussian(dist, sigma_s);
                float gr = gaussian(colorDistance, sigma_r);
                float weight = gs * gr;

                iFiltered += image_buffer[neighbor_id] * weight;
                wP += weight;
            }
        }
    }

    return wP > 0.0f ? iFiltered / wP : float(image_buffer[pixel_id]);
}

__device__ unsigned char d_clamp_pixel_value(float pixel) {
    return pixel > 255 ? 255 : pixel < 0 ? 0 : static_cast<unsigned char>(pixel);
}

__global__ void apply_bilateral_filter_kernel(unsigned char* input_buffer,
                                              unsigned char* filtered_image, 
                                              int width, int height, int num_channels,
                                              float sigma_s, float sigma_r) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= 1 && x < width - 1 && y >= 1 && y < height - 1) {
        for (int c = 0; c < num_channels; c++) {
            int idx = (y * width + x) * num_channels + c;
            float filtered_val = d_bilateral_filter(input_buffer, x, y, width, height, num_channels, sigma_s, sigma_r, idx);
            filtered_image[idx] = d_clamp_pixel_value(filtered_val);
        }
    }
}

int main(int argc, char** argv) {
    if (argc != 3) {
        std::cerr << "Invalid argument, should be: ./executable /path/to/input/jpeg /path/to/output/jpeg\n";
        return -1;
    }

    const char* input_filename = argv[1];
    const char* output_filename = argv[2];
    std::cout << "Input file from: " << input_filename << "\n";

    // 读取JPEG图像
    auto input_jpeg = read_from_jpeg(input_filename);
    if (input_jpeg.buffer == nullptr) {
        std::cerr << "Failed to read input JPEG image\n";
        return -1;
    }

    int width = input_jpeg.width;
    int height = input_jpeg.height;
    int num_channels = input_jpeg.num_channels;
    size_t buffer_size = width * height * num_channels;

    unsigned char* filteredImage = new unsigned char[buffer_size];

    // 分配GPU内存
    unsigned char* d_input_buffer;
    unsigned char* d_filtered_image;

    hipMalloc((void**)&d_input_buffer, buffer_size);
    hipMalloc((void**)&d_filtered_image, buffer_size);

    hipMemset(d_filtered_image, 0, buffer_size);

    // 将数据从主机传输到设备
    hipMemcpy(d_input_buffer, input_jpeg.buffer, buffer_size, hipMemcpyHostToDevice);

    // 设置CUDA网格和块大小
    dim3 blockDim(32, 32);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x,
                 (height + blockDim.y - 1) / blockDim.y);

    // 记录CUDA事件
    hipEvent_t start, stop;
    float gpuDuration;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // 执行双边滤波
    hipEventRecord(start, 0); // GPU开始时间
    apply_bilateral_filter_kernel<<<gridDim, blockDim>>>(d_input_buffer, d_filtered_image, width, height, num_channels, SIGMA_S, SIGMA_R);
    hipEventRecord(stop, 0); // GPU结束时间
    hipEventSynchronize(stop);

    // 获取GPU计算时间
    hipEventElapsedTime(&gpuDuration, start, stop);

    // 将数据从设备传输回主机
    hipMemcpy(filteredImage, d_filtered_image, buffer_size, hipMemcpyDeviceToHost);

    // 保存JPEG输出图像
    JPEGMeta output_jpeg{filteredImage, width, height, num_channels, input_jpeg.color_space};
    if (export_jpeg(output_jpeg, output_filename)) {
        std::cerr << "Failed to write output JPEG\n";
        return -1;
    }

    // 清理内存
    hipFree(d_input_buffer);
    hipFree(d_filtered_image);
    delete[] input_jpeg.buffer;
    delete[] filteredImage;

    std::cout << "Transformation Complete!" << std::endl;
    std::cout << "GPU Execution Time: " << gpuDuration << " milliseconds" << std::endl;

    // 销毁CUDA事件
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
