#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <chrono>
#include <hip/hip_runtime.h>
#include "utils.hpp"  // Make sure this path is correct

#define FILTER_RADIUS 2
#define SIGMA_S 10.0f
#define SIGMA_R 50.0f

__device__ float gaussian(float x, float sigma) {
    return expf(-0.5f * x * x / (sigma * sigma));
}

__device__ float d_bilateral_filter(unsigned char* image_buffer,
                                    int x, int y, int width, int height, int channels,
                                    float sigma_s, float sigma_r, int pixel_id) {
    float iFiltered = 0.0f;
    float wP = 0.0f;
    
    for (int dy = -FILTER_RADIUS; dy <= FILTER_RADIUS; dy++) {
        for (int dx = -FILTER_RADIUS; dx <= FILTER_RADIUS; dx++) {
            int nx = x + dx;
            int ny = y + dy;

            if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
                int neighbor_id = (ny * width + nx) * channels + pixel_id % channels;
                float dist = sqrtf(dx * dx + dy * dy);
                float colorDistance = fabs(float(image_buffer[neighbor_id]) - float(image_buffer[pixel_id]));

                float gs = gaussian(dist, sigma_s);
                float gr = gaussian(colorDistance, sigma_r);
                float weight = gs * gr;

                iFiltered += image_buffer[neighbor_id] * weight;
                wP += weight;
            }
        }
    }

    return wP > 0.0f ? iFiltered / wP : float(image_buffer[pixel_id]);
}

__device__ unsigned char d_clamp_pixel_value(float pixel) {
    return pixel > 255 ? 255 : pixel < 0 ? 0 : static_cast<unsigned char>(pixel);
}

__global__ void apply_bilateral_filter_kernel(unsigned char* input_buffer,
                                              unsigned char* filtered_image, 
                                              int width, int height, int num_channels,
                                              float sigma_s, float sigma_r) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= 1 && x < width - 1 && y >= 1 && y < height - 1) {
        for (int c = 0; c < num_channels; c++) {
            int idx = (y * width + x) * num_channels + c;
            float filtered_val = d_bilateral_filter(input_buffer, x, y, width, height, num_channels, sigma_s, sigma_r, idx);
            filtered_image[idx] = d_clamp_pixel_value(filtered_val);
        }
    }
}

int main(int argc, char** argv) {
    if (argc != 3) {
        std::cerr << "Invalid argument, should be: ./executable /path/to/input/jpeg /path/to/output/jpeg\n";
        return -1;
    }

    const char* input_filename = argv[1];
    const char* output_filename = argv[2];
    std::cout << "Input file from: " << input_filename << "\n";

    JPEGMeta input_jpeg = read_from_jpeg(input_filename);
    if (input_jpeg.buffer == nullptr) {
        std::cerr << "Failed to read input JPEG image\n";
        return -1;
    }

    int width = input_jpeg.width;
    int height = input_jpeg.height;
    int num_channels = input_jpeg.num_channels;
    size_t buffer_size = width * height * num_channels;

    unsigned char* d_input_buffer;
    unsigned char* d_filtered_image;
    hipMalloc((void**)&d_input_buffer, buffer_size);
    hipMalloc((void**)&d_filtered_image, buffer_size);

    hipMemcpy(d_input_buffer, input_jpeg.buffer, buffer_size, hipMemcpyHostToDevice);

    dim3 blockDim(32, 32);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x,
                 (height + blockDim.y - 1) / blockDim.y);

    apply_bilateral_filter_kernel<<<gridDim, blockDim>>>(d_input_buffer, d_filtered_image, width, height, num_channels, SIGMA_S, SIGMA_R);

    unsigned char* filteredImage = new unsigned char[buffer_size];
    hipMemcpy(filteredImage, d_filtered_image, buffer_size, hipMemcpyDeviceToHost);

    // Wrapping filtered image data into JPEGMeta structure for export
    JPEGMeta output_jpeg{
        filteredImage,
        width,
        height,
        num_channels,
        input_jpeg.color_space
    };

    if (export_jpeg(output_jpeg, output_filename) != 0) {
        std::cerr << "Failed to write output JPEG\n";
        return -1;
    }

    hipFree(d_input_buffer);
    hipFree(d_filtered_image);
    delete[] input_jpeg.buffer;
    delete[] filteredImage;

    std::cout << "Filtering complete and image saved to " << output_filename << std::endl;
    return 0;
}
